#include "hip/hip_runtime.h"
#include "cutil_inline.h"

#include "math.h"
#include "time.h"
#include "stdlib.h"
#include "limits.h"
#include "stdio.h"

#define ITERS 10

// Direction of price for the given manufacturer.
// Up means prices are increasing, down is decreasing
#define NUM_STRATEGIES 2
#define STRATEGY_UP 0
#define STRATEGY_DOWN 1

//#define BLOCK_SIZE 32
//#define GRID_SIZE 32
#define NUM_MANUFACTURERS 2
#define NUM_CONSUMERS 100
#define MAX_MARGINAL 250
#define BASE_INCOME 20000
#define PRICE_INCREMENT 5
// The price of any product cannot exceed this value multiplied by the marginal
// cost for that product.
#define MAX_PRICE_MULTIPLIER 5.0f 

// The gradient/decay rate of the function used to determine
// fitness for roulette-wheel selection, which is used to
// find the manufacturer to buy from
#define LOYALTY_ALPHA 8.0f

// By how much we multiply the score of the preferred manufacturer
#define LOYALTY_MULTIPLIER 2.0f

// What additional price over the cheapest we are willing to consider.
// E.g. 0.5 means we never buy products 50% more expensive than cheapest
#define RIPOFF_MULTIPLIER 1.0f

// Whether the consumers choose which product to buy based on loyalty.
// Otherwise, they just pick the cheapest
#define LOYALTY_ENABLED 0

const char* products[] = {"milk"};//, "bread", "toilet_paper", "butter", "bacon", "cheese"};
int NUM_PRODUCTS = sizeof(products)/sizeof(char*);

// Arrays mapping manufacturer ID to profit on each day
typedef struct
{
  int* two_days_ago;
  int* yesterday;
  int* today;
} profits;

int select_loyalty();
double gaussrand();
double positive_gaussrand();
void print_array(float*, unsigned int);
void print_2d_array(float** data_in, unsigned int size1, unsigned int size2);
void print_2d_int_array(int** data_in, unsigned int size1, unsigned int size2);
void print_2d_1d_int_array(int* data_in, unsigned int size1, unsigned int size2);
void print_int_array(int* data_in, unsigned int size);
int* calculate_num_purchases(int* purchases, unsigned int num_consumers);
void print_profit_struct(profits* profit, unsigned int num_manufacturers);
int get_max_ind(int* array, unsigned int size);
int get_min_ind(int* array, unsigned int size);
void put_plot_line(FILE* fp, int* arr, unsigned int size, int x);
void modify_price(int* marginal_cost, int* max_cost, int manufacturer_id, int product_id, int strategy, int* price_arr, int num_manufacturers);
int* manufacturer_loyalty_counts(int* loyal_arr, int num_manufacturers, int num_consumers);
__device__ int d_get_max_ind(int* array, unsigned int size);
__global__ void d_update_loyalties(int* choices, int* loyalties, unsigned int num_manufacturers,
                                   unsigned int num_customers);
void launch_update_loyalties(int* choices, int* loyalties, unsigned int num_consumers,
                             unsigned int num_manufacturers);

// dim1 = first dimension, dim2 is second
// So to do arr[1][5] -> idx(1, 5, width)
int idx(unsigned int dim1, unsigned int dim2, unsigned int width)
{
  if (dim2 >= width) 
  {
    fprintf(stderr, "Error! IndexOutOfBounds. dim2=%d, width=%d. Exiting...\n", dim2, width);
    exit(-1);
  }
  
  return dim2 + dim1*width;
}

int val(int* array, unsigned int dim1, unsigned int dim2, unsigned int width) 
{
  return array[idx(dim1,dim2,width)];
}

void set_val(int* array, unsigned int dim1, unsigned int dim2, unsigned int width, int val) 
{
  array[idx(dim1,dim2,width)] = val;
}

// Each manufacturer has a strategy at a given moment in time.
// Either they are raising their profits or decreasing them. Here, we initialise
// these values to random strategies for the first time step
int* init_strategy() 
{
  int* price_strategy = (int*) malloc(NUM_MANUFACTURERS*sizeof(int));
  
  int i;
  for (i = 0; i < NUM_MANUFACTURERS; i++) 
  {
    // Randomly choose int between 0 and num of strategies-1
    float randVal = (float)rand()/RAND_MAX;
    price_strategy[i] = (int)(randVal*NUM_STRATEGIES);
  }
  return price_strategy;
}

// Initialises the marginal and maximum costs for each product. The maximum price is
// some multiple of the marginal cost.
int* init_marginal_cost()
{
    int* marginal_cost = (int*) malloc(NUM_PRODUCTS * sizeof(int));

    int i;
    
    for (i = 0; i < NUM_PRODUCTS; ++i) {
        float rval = (float)rand()/RAND_MAX;
        //marginal_cost[i] = (int)(rval * MAX_MARGINAL);
        marginal_cost[i] = 100+(i*10);
    
//    printf("Marginal cost for %s is %d.\n", products[i], marginal_cost[i]);
    }
    return marginal_cost;
}

// Initialises the maximum costs of products based on their marginal cost
int* init_max_cost(int* marginal_cost){
    int* max_cost = (int*) malloc(NUM_PRODUCTS * sizeof(int));

    int i;

    for (i = 0; i < NUM_PRODUCTS; ++i) {
        max_cost[i] = MAX_PRICE_MULTIPLIER * marginal_cost[i];
    }
    return max_cost; 
}

// Rand*MC*3 (roughly)
// Initialises the prices for each product.
int* init_prices(int* marginal_cost)
{
  int i;
  int j;

  int* price = (int*) malloc(NUM_PRODUCTS * NUM_MANUFACTURERS * sizeof(int));
  const int width = NUM_MANUFACTURERS;

  for (i = 0; i < NUM_PRODUCTS; ++i) {
    for (j = 0; j < NUM_MANUFACTURERS; ++j) {
      float rval = (float)rand()/RAND_MAX;
      float val = marginal_cost[i] + (rval * marginal_cost[i]);
      set_val(price, i, j, width, val);
//      printf("Price for %s (%d) from manufacturer %d: %d\n", products[i], i, j, price[i][j]);
    }
  }
  return price;
}

// Uniformly distributed
int* init_loyalty()
{
    int* loyalty = (int*) malloc(NUM_CONSUMERS * sizeof(int));
    
    int i;
    int* counts = (int*) malloc(NUM_MANUFACTURERS * sizeof(int));
    
    for (i = 0; i < NUM_CONSUMERS; ++i) {
        loyalty[i] = select_loyalty();
        //	printf("Customer %d loyal to manufacturer %d\n", i, loyalty[i]);
        counts[loyalty[i]]++;
    }

    /* for (i = 0; i < NUM_MANUFACTURERS; ++i) { */
    /*     printf("Manufacturer %d has %d loyal customers.\n", i, counts[i]); */
    /* } */
    return loyalty;
}

// Returns uniform random number in the range [0, NUM_MANUFACTURERS]
int select_loyalty()
{
  int i;
  float rval = (float)rand()/RAND_MAX;

  float split = 1.0/NUM_MANUFACTURERS;
  for (i = 0; i < NUM_MANUFACTURERS; ++i) {
    if (rval < split * (i + 1))
      return i;
  }
  return i;
}

/*
 * Gaussian over population. Currently generates values using a gaussian tail
 * distribution - there will be a lot of people who have an income around 
 * the base income, and fewer with higher incomes.
 */
int* init_income()
{
    int* income = (int*) malloc(NUM_CONSUMERS * sizeof(int));
    
    int i;
    for (i = 0; i < NUM_CONSUMERS; ++i) {
        income[i] = BASE_INCOME * (positive_gaussrand() + 1);
        printf("Income of household %d: %d\n", i, income[i]);
    }
    return income;
}

// Initialise last two days of profits with fake values.
// All profits two days ago are set to 0 and for yesterday 
// are set to 1. Thus, all profits increase so currently
// active strategies are kept in place and acted on
profits* init_profits() 
{
    profits* profit_history = (profits*) malloc(sizeof(profits));
    profit_history->two_days_ago = (int*) malloc(sizeof(int)*NUM_MANUFACTURERS);
    profit_history->yesterday = (int*) malloc(sizeof(int)*NUM_MANUFACTURERS);
    profit_history->today = (int*) malloc(sizeof(int)*NUM_MANUFACTURERS);

    int man;
    for (man = 0; man < NUM_MANUFACTURERS; ++man)
    {
        profit_history->two_days_ago[man] = 0;
        profit_history->yesterday[man] = 1;
    }
    return profit_history;
}



/* Generate a gaussian random value in the interval [0,infinity] */
double positive_gaussrand()
{
    double r;
    while ((r = gaussrand()) < 0);
    return r;
}

// Polar method implementation taken from c-faq.com/lib/gaussian.html
double gaussrand()
{
  static double V1, V2, S;
  static int phase = 0;
  double X;
    
  if (phase == 0){
    do {
      double U1 = (double)rand()/RAND_MAX;
      double U2 = (double)rand()/RAND_MAX;
	    
      V1 = 2 * U1 - 1;
      V2 = 2 * U2 - 1;
      S = V1 * V1 + V2 * V2;
    } while (S >= 1 || S == 0);
    X = V1 * sqrt(-2 * log(S) / S);
  } else {
    X = V2 * sqrt(-2 * log(S) / S);
  }

  phase = 1 - phase;
    
  return X;
}

// Get the manufacturer ID fom which the consumer chooses to 
// purchase the given product
int host_consumer_choice(int* loyalty, int* price, int consumer_id, int product_id, int cheapest_man, int loyalty_enabled, int num_manufacturers) {
  if (! loyalty_enabled) 
  {
    return cheapest_man;
  }
  
  // If cheapest manufacturer is already preferred, pick that
  if (loyalty[consumer_id] == cheapest_man) 
  {
    //   printf("Preferred is cheapest. Returning %d\n",cheapest_man);
    return cheapest_man;
  }
  else
  {
    float cheapest_price = (float) val(price, product_id, cheapest_man, num_manufacturers);
    float* scores = (float*) malloc(sizeof(float)*num_manufacturers);

    float total_score = 0.0f;
    
    for (int man = 0; man < num_manufacturers; man++) 
    {
      // equiv. of x in function
      int price_diff = val(price, product_id, man, num_manufacturers) - cheapest_price;
      float score;
      if (price_diff > RIPOFF_MULTIPLIER*cheapest_price) 
      {
        score = 0;
      }
      else
      {
        score = cheapest_price/(LOYALTY_ALPHA*price_diff + cheapest_price);
        total_score += score;
      }

      if (man == loyalty[consumer_id])
      {
        score *= LOYALTY_MULTIPLIER;
      }

      scores[man] = score;
    }

    float ran = (float)rand() / RAND_MAX * total_score;
    float score_so_far = 0.0f;


    printf("Scores array: ");
    print_array(scores, num_manufacturers);
    printf("Rand is %.5f\n", ran);

    for (int man = 0; man < num_manufacturers; man++) 
    {
      score_so_far += scores[man];
      if (score_so_far >= ran)
      {
        return man;
      }
    }
  }
  
  // Should have returned by now, so return -1 to crash or segfault or something
  fprintf(stderr, "Error! Didn't select anything in roulette wheel selection inside "\
          "host_consumer_choice. Exiting...\n");
  exit(1);
  return -1;
}

// Get tomorrow's price for the given product ID
void host_price_response(int* marginal_cost, int* max_cost, profits* profit_history, int manufacturer_id, int product_id, int* price_strategy_arr, int* price_arr, int num_manufacturers) {
  int current_strategy = price_strategy_arr[manufacturer_id];
  int profit1 = profit_history->two_days_ago[manufacturer_id];
  int profit2 = profit_history->yesterday[manufacturer_id];

  // If profit decreased, switch strategy
  if (profit1 > profit2) 
  {
    if (current_strategy == STRATEGY_UP) 
    {
      price_strategy_arr[manufacturer_id] = STRATEGY_DOWN;
    }
    else
    {
      price_strategy_arr[manufacturer_id] = STRATEGY_UP;
    }
  }
  else if (profit1 == profit2) {
    price_strategy_arr[manufacturer_id] = STRATEGY_DOWN;
  }

  modify_price(marginal_cost, max_cost, manufacturer_id, product_id, price_strategy_arr[manufacturer_id], price_arr, num_manufacturers);
}

// Modifies the price the manufacturer charges for the given product based on
// the current strategy. The price can never exceed some multiple of the marginal
// cost, and can never fall below the marginal cost.
void modify_price(int* marginal_cost, int* max_cost, int manufacturer_id, int product_id, int strategy, int* price_arr, int num_manufacturers)
{
  int price_of_prod = val(price_arr, product_id, manufacturer_id, num_manufacturers);
  
  if (strategy == STRATEGY_UP && price_of_prod <= max_cost[product_id] - PRICE_INCREMENT) 
  {
    int new_price = price_of_prod + PRICE_INCREMENT;
    set_val(price_arr, product_id, manufacturer_id, num_manufacturers, new_price);
  }
  else if (strategy == STRATEGY_DOWN && price_of_prod >= marginal_cost[product_id] + PRICE_INCREMENT) 
  {
    int new_price = price_of_prod - PRICE_INCREMENT;
    set_val(price_arr, product_id, manufacturer_id, num_manufacturers, new_price);
  }
}

// Gets the ID of the manufacturer which has the cheapest product for the given ID.
int get_cheapest_man(int* price, int product_id)
{
  int* price_arr_point = &price[product_id*NUM_MANUFACTURERS];
  return get_min_ind(price_arr_point, NUM_MANUFACTURERS);
}

// We pass in the array of integers containing which manufacturer each
// consumer chooses based on the host_consumer_choice function. The return
// is an array containing the number of purchases made for each manufacturer
int* calculate_num_purchases(int* purchases, unsigned int num_consumers,
                             unsigned int num_manufacturers){
  int* counts = (int*)calloc(num_manufacturers, sizeof(int));
  for (int consumer_num = 0; consumer_num < num_consumers; consumer_num++){
    counts[purchases[consumer_num]]++;
  }

  return counts;
}

// Adds the profit for the given product into the profit_today array. The array
// should be initialised with zeroes on the first call
void profit_for_product(int* purchases, int* profit_today, int* price,
                        int marginal_cost, unsigned int num_manufacturers){
  for (int man_id = 0; man_id < num_manufacturers; man_id++){
    profit_today[man_id] += purchases[man_id] * (price[man_id] - marginal_cost);
  }
}

// Shifts pointers around in the given profit struct so that today's profits are
// yesterdays, and yesterday's are the profits two days ago. The array used to
// store the profits two days ago is zeroed and set to be used by the today array
void swap_profit_pointers(profits* profit, unsigned int num_manufacturers)
{
  int* tmp = profit->two_days_ago;
  profit->two_days_ago = profit->yesterday;
  profit->yesterday = profit->today;
  profit->today = tmp;
  bzero(profit->today, sizeof(int) * num_manufacturers);
}

// Update the loyalties of customers based on the number of purchases
// made from each manufacturer during the last day
void update_loyalties(int* choices, int* loyalties, unsigned int num_consumers,
                      unsigned int num_manufacturers)
{
    for (int cons_id = 0; cons_id < num_consumers; cons_id++)
    {
        // Most purchased-from manufacturer
        int* choices_subarr = &choices[cons_id*num_manufacturers];
        int most_purchased = get_max_ind(choices_subarr, num_manufacturers);

        // If we purchase more products from a manufacturer to whom we are
        // not currently loyal to, we switch to the one that we purchased
        // most from.
        if (val(choices, cons_id, most_purchased, num_manufacturers) !=
            val(choices, cons_id, loyalties[cons_id], num_manufacturers)) {
            loyalties[cons_id] = most_purchased;
        }
    }
}

  /* printf("Creating arrays and references.\n"); */
  /* int host_cust[6] = {5,10,5,10,7,10}; */
  /* int host_loyalty[] = {0,0,0}; */
  /* int cust_memsize = sizeof(host_cust);//sizeof(int*) * 3 + sizeof(int) * 2 * 3; */
  /* int loyalty_memsize = sizeof(host_loyalty);//3*sizeof(int); */
  /* int* host_cust_res = (int*) malloc(cust_memsize); */
  /* int* host_loyalty_res = (int*) malloc(loyalty_memsize); */
  /* int* dev_cust; */
  /* int* dev_loyalty; */
  /* printf("Allocating device memory\n"); */
  /* cutilSafeCall(hipMalloc((void**) &dev_cust, cust_memsize)); */
  /* cutilSafeCall(hipMalloc((void**) &dev_loyalty, loyalty_memsize)); */
  /* cutilSafeCall(hipMemcpy(dev_cust, host_cust, cust_memsize, hipMemcpyHostToDevice)); */
  /* cutilSafeCall(hipMemcpy(dev_loyalty, host_loyalty, loyalty_memsize, hipMemcpyHostToDevice)); */
  /* printf("Got to the kernel call\n"); */
  /* print_int_array(host_loyalty, 3); */
  /* d_update_loyalties<<<1, 3>>>(dev_cust, dev_loyalty, 2, 3); */

  /* /\* int* dev_loyalty_res; *\/ */
  /* /\* int** dev_cust_out; *\/ */
  /* /\* cutilSafeCall(hipMalloc((void***) &dev_cust_out, cust_memsize)); *\/ */
  /* /\* cutilSafeCall(hipMalloc((void**) &dev_loyalty_res, loyalty_memsize)); *\/ */

  /* printf("Device call finished. Copying data from dev to host...\n"); */

  /* cutilSafeCall(hipMemcpy(host_loyalty_res, dev_loyalty, */
  /*                          loyalty_memsize, hipMemcpyDeviceToHost)); */
  /* /\* cutilSafeCall(hipMemcpy(host_cust_res, dev_cust,  *\/ */
  /* /\*                          cust_memsize, hipMemcpyDeviceToHost)); *\/ */


  /* printf("\n\n"); */
  /* print_int_array(host_loyalty_res, 3); */

// Performs the necessary memory allocations and conversions and launches the
// kernel function to compute the updated loyalties.
void launch_update_loyalties(int* choices, int* loyalties, unsigned int num_consumers,
                             unsigned int num_manufacturers)
{
    int nblocks = 1, nthreads = num_consumers;
    int choice_memsize = num_consumers * num_manufacturers * sizeof(int);
    int loyalty_memsize = num_consumers * sizeof(int);
    int* dev_choices;
    int* dev_loyalty;
    
    // Allocate device memory for both arrays
    cutilSafeCall(hipMalloc((void**) &dev_choices, choice_memsize));
    cutilSafeCall(hipMalloc((void**) &dev_loyalty, loyalty_memsize));

    // Copy the data into the device arrays. Only need to do this for the choices, since that
    // is the only data which is read - the device will overwrite values in the loyalties array.
    cutilSafeCall(hipMemcpy(dev_choices, choices, choice_memsize, hipMemcpyHostToDevice));
//    cutilSafeCall(hipMemcpy(dev_loyalty, loyalties, loyalty_memsize, hipMemcpyHostToDevice));

//    print_int_array(host_loyalty, 3);
    d_update_loyalties<<<nblocks, nthreads>>>(dev_choices, dev_loyalty, num_manufacturers, num_consumers);

    cutilSafeCall(hipMemcpy(loyalties, dev_loyalty, loyalty_memsize, hipMemcpyDeviceToHost));

//    print_int_array(host_loyalty_res, 3);
}

// Updates the loyalties of each customer after the purchases for the day have been made.
// The number of threads should be the number of consumers.
__global__ void d_update_loyalties(int* choices, int* loyalties, unsigned int num_manufacturers,
                                   unsigned int num_customers)
{
    int tid = threadIdx.x + blockDim.x*blockIdx.x;

    loyalties[tid] = d_get_max_ind(choices + tid * num_manufacturers, num_manufacturers);
}

// Get the index of the maximum value in the given array.
__device__ int d_get_max_ind(int* array, unsigned int size)
{
  int best = 0;
  for (int i = 1; i < size; i++)
  {
    if (array[i] > array[best])
    {
      best = i;
    }
  }
  return best;
}

// Get tomorrow's strategy for each manufacturer
// This sets strategy only. Price needs to be set separately.
// Number of threads should be num of manufacturers
__global__ void device_price_response(int* price_strategy,
                                      int* profit_two_days_ago, 
                                      int* profit_yesterday) {
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int manufacturer_id = tid;

  int current_strategy = price_strategy[manufacturer_id];
  int profit1 = profit_two_days_ago[manufacturer_id];
  int profit2 = profit_yesterday[manufacturer_id];

  // If profit decreased, switch strategy
  if (profit1 > profit2) 
  {
    if (current_strategy == STRATEGY_UP) 
    {
      price_strategy[manufacturer_id] = STRATEGY_DOWN;
    }
    else
    {
      price_strategy[manufacturer_id] = STRATEGY_UP;
    }
  }
  else if (profit1 == profit2) {
    price_strategy[manufacturer_id] = STRATEGY_DOWN;
  }
}

void launch_device_price_response(int* price_strategy,
                                  int* profit_two_days_ago, 
                                  int* profit_yesterday,
                                  int num_manufacturers)
{
  int blocks = 1;
  int threadsPerBlock = num_manufacturers;
  
  int* dev_price_strategy;
  int* dev_profit_two_days_ago;
  int* dev_profit_yesterday;
  int mem_size = num_manufacturers * sizeof(int);
  
  cutilSafeCall(hipMalloc((void**) &dev_price_strategy, mem_size));
  cutilSafeCall(hipMalloc((void**) &dev_profit_two_days_ago, mem_size));
  cutilSafeCall(hipMalloc((void**) &dev_profit_yesterday, mem_size));

  cutilSafeCall(hipMemcpy(dev_price_strategy, price_strategy, mem_size, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dev_profit_two_days_ago, profit_two_days_ago, mem_size, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dev_profit_yesterday, profit_yesterday, mem_size, hipMemcpyHostToDevice));

  device_price_response<<<blocks, threadsPerBlock>>>(price_strategy,
                                                     profit_two_days_ago,
                                                     profit_yesterday);

  cutilSafeCall(hipMemcpy(price_strategy, dev_price_strategy, mem_size, hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(profit_two_days_ago, dev_profit_two_days_ago, mem_size, hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(profit_yesterday, dev_profit_yesterday, mem_size, hipMemcpyDeviceToHost));
}


// Modifies the price the manufacturer charges for each product based on
// the current strategy. The price can never exceed some multiple of the marginal
// cost, and can never fall below the marginal cost.
// Number of threads should be num_manufacturers*num_products
__global__ void device_modify_price(int* strategy_arr, 
                                    int** price_arr, 
                                    int* max_cost_arr,
                                    int* marginal_cost_arr,
                                    int num_manufacturers,
                                    int num_products)
{
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int manufacturer_id = tid / num_manufacturers;
  const int product_id = tid % num_manufacturers;
  
  if (strategy_arr[manufacturer_id] == STRATEGY_UP 
      && price_arr[product_id][manufacturer_id] <= max_cost_arr[product_id] - PRICE_INCREMENT) 
  {
    price_arr[product_id][manufacturer_id] += PRICE_INCREMENT;
  }
  else if (strategy_arr[manufacturer_id] == STRATEGY_DOWN 
           && price_arr[product_id][manufacturer_id] >= marginal_cost_arr[product_id] + PRICE_INCREMENT)
  {
    price_arr[product_id][manufacturer_id] -= PRICE_INCREMENT;
  }
}

void launch_device_modify_price(int* strategy_arr, 
                                int** price_arr, 
                                int* max_cost_arr,
                                int* marginal_cost_arr,
                                int num_manufacturers,
                                int num_products)
{
  int blocks = 1;
  int threadsPerBlock = num_manufacturers*num_products;
  
  // Mem size for arrays containing elements up to num_manufacturers 
  int man_mem_size = num_manufacturers * sizeof(int);

  // Mem size for arrays containing elements up to num_products
  int prod_mem_size = num_products * sizeof(int);

  // Mem size for arrays containing elements of num_products*num_manufacturers
  int man_prod_mem_size = num_manufacturers * prod_mem_size;

  int* dev_strategy_arr;
  int** dev_price_arr; // 2D array needs conversion to 1D, as well as calls below
  int* dev_max_cost_arr;
  int* dev_marginal_cost_arr;
  
  cutilSafeCall(hipMalloc((void**) &dev_strategy_arr, man_mem_size));
  cutilSafeCall(hipMalloc((void**) &dev_price_arr, man_prod_mem_size));
  cutilSafeCall(hipMalloc((void**) &dev_max_cost_arr, prod_mem_size));
  cutilSafeCall(hipMalloc((void**) &dev_marginal_cost_arr, prod_mem_size));

  cutilSafeCall(hipMemcpy(dev_strategy_arr, strategy_arr, man_mem_size, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dev_price_arr, price_arr, man_prod_mem_size, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dev_max_cost_arr, max_cost_arr, prod_mem_size, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dev_marginal_cost_arr, marginal_cost_arr, prod_mem_size, hipMemcpyHostToDevice));

  device_modify_price<<<blocks, threadsPerBlock>>>(dev_strategy_arr,
                                                   dev_price_arr, 
                                                   dev_max_cost_arr,
                                                   dev_marginal_cost_arr,
                                                   num_manufacturers,
                                                   num_products);

  cutilSafeCall(hipMemcpy(strategy_arr, dev_strategy_arr, man_mem_size, hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(price_arr, dev_price_arr, man_prod_mem_size, hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(max_cost_arr, dev_max_cost_arr, prod_mem_size, hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(marginal_cost_arr, dev_marginal_cost_arr, prod_mem_size, hipMemcpyDeviceToHost));
}


void host_equilibriate(int* price, int* loyalty,
                       profits* profit, int* price_strategy,
                       int* marginal_cost, int* max_cost, int days, int loyalty_enabled, 
                       char* profitFilename, char* priceFilename,
                       char* loyaltyFilename)
{
  int day_num;
  int man_id, prod_id, cons_id;
  
  FILE* profitFile = fopen(profitFilename, "w");
  FILE* priceFile = fopen(priceFilename, "w");
  FILE* loyalFile = fopen(loyaltyFilename, "w");
  
  for (day_num = 0; day_num < days; day_num++)
  {
    printf("Old prices (line = product):\n");
    print_2d_1d_int_array(price, NUM_PRODUCTS, NUM_MANUFACTURERS);

    printf("Strategies (0 is up, 1 is down): \n");
    print_int_array(price_strategy, NUM_MANUFACTURERS);
    
    for (man_id = 0; man_id < NUM_MANUFACTURERS; man_id++){
      for (prod_id = 0; prod_id < NUM_PRODUCTS; prod_id++){
        host_price_response(marginal_cost, max_cost, profit, man_id, prod_id, price_strategy, price, NUM_MANUFACTURERS);
      }
    }

    printf("New prices (line = product):\n");
    print_2d_1d_int_array(price, NUM_PRODUCTS, NUM_MANUFACTURERS);


    int* picks = (int*)malloc(sizeof(int) * NUM_CONSUMERS);
    // This array contains the number of picks that a consumer has made from
    // each manufacturer. The first dimension is the consumer id, and the second
    // is the manufacturer.
    int* cons_choices = (int*) calloc(NUM_CONSUMERS * NUM_MANUFACTURERS, sizeof(int));
    /* for (int i = 0; i < NUM_CONSUMERS; i++){ */
    /*   cons_choices[i] = (int*) calloc(sizeof(int), NUM_MANUFACTURERS); */
    /* } */

    printf("Printing profits for man=0\n");
    
    for (prod_id = 0; prod_id < NUM_PRODUCTS; prod_id++){
        int cheapest = get_cheapest_man(price, prod_id);
      // TODO: Calculate the scores for this product here, rather than multiple times
      // in the consumer choice function.
      for (cons_id = 0; cons_id < NUM_CONSUMERS; cons_id++){
        picks[cons_id] = host_consumer_choice(loyalty, price, cons_id, prod_id, cheapest, loyalty_enabled, NUM_MANUFACTURERS);
        // Increment the number of times the consumer picked the manufacturer returned from
        // the host_consumer_choice function
        int new_val = val(cons_choices, cons_id, picks[cons_id], NUM_MANUFACTURERS) + 1;
        set_val(cons_choices, cons_id, picks[cons_id], NUM_MANUFACTURERS, new_val);
        //cons_choices[cons_id][picks[cons_id]]++;
      }
      int* counts = calculate_num_purchases(picks, NUM_CONSUMERS, NUM_MANUFACTURERS);
      printf("Number of purchases for each product:\n");
      print_int_array(counts, NUM_MANUFACTURERS);

      printf("ProfitToday before prod %d: %d\n", prod_id, profit->today[0]);
      
      int* price_arr_point = &price[prod_id*NUM_MANUFACTURERS];
      
      profit_for_product(counts, profit->today, price_arr_point, marginal_cost[prod_id], NUM_MANUFACTURERS);
      print_profit_struct(profit, NUM_MANUFACTURERS);
    }

    update_loyalties(cons_choices, loyalty, NUM_CONSUMERS, NUM_MANUFACTURERS);
    printf("Loyalties:\n");
    print_int_array(loyalty, NUM_CONSUMERS);
    printf("Printing cons choices.\n");
    print_2d_1d_int_array(cons_choices, NUM_CONSUMERS, NUM_MANUFACTURERS);

    put_plot_line(profitFile, profit->today, NUM_MANUFACTURERS, day_num);
    int prod_to_print = 0;
    int* price_arr_point = &price[prod_to_print*NUM_MANUFACTURERS];
    put_plot_line(priceFile, price_arr_point, NUM_MANUFACTURERS, day_num);
    int* ct = manufacturer_loyalty_counts(loyalty, NUM_MANUFACTURERS, NUM_CONSUMERS);
    put_plot_line(loyalFile, ct, NUM_MANUFACTURERS, day_num);
    // swap the pointers inside the profit struct so that we can overwrite without needing to free
    swap_profit_pointers(profit, NUM_MANUFACTURERS);

    printf("A new day dawns.\n\n\n\n\n\n");
  }

  fclose(profitFile);
  fclose(priceFile);
  fclose(loyalFile);
}

int* manufacturer_loyalty_counts(int* loyal_arr, int num_manufacturers, int num_consumers)
{
  int* counts = (int*)calloc(num_manufacturers, sizeof(int));
  for (int consumer_num = 0; consumer_num < num_consumers; consumer_num++){
    counts[loyal_arr[consumer_num]]++;
  }
  return counts;
}

// Writes the given array into the provided file pointer. The x value
// is printed before the values in the array.
void put_plot_line(FILE* fp, int* arr, unsigned int size, int x)
{
  fprintf(fp, "%d", x);

  for (int i = 0; i < size; i++)
  {
    fprintf(fp, " %d", arr[i]);
  }

  fprintf(fp, "\n");
}

int get_min_ind(int* array, unsigned int size)
{
  int best = 0;
  for (int i = 1; i < size; i++)
  {
    if (array[i] < array[best])
    {
      best = i;
    }
  }
  return best;
}

int get_max_ind(int* array, unsigned int size)
{
  int best = 0;
  for (int i = 1; i < size; i++)
  {
    if (array[i] > array[best])
    {
      best = i;
    }
  }
  return best;
}

void copy_array(float* from, float* to, unsigned int size) {
  for (int i = 0; i < size; i++) {
    to[i] = from[i];
  }
}

void print_array(float* data_in, unsigned int size)
{
  for (int i=0; i < size-1; i++) {
    printf("%f,", data_in[i]);
  }
  printf("%f\n", data_in[size-1]);
}

void print_profit_struct(profits* profit, unsigned int num_manufacturers)
{
  printf("Profits\nTwo days ago: ");
  print_int_array(profit->two_days_ago, num_manufacturers);
  printf("Yesterday ");
  print_int_array(profit->yesterday, num_manufacturers);
  printf("Today ");
  print_int_array(profit->today, num_manufacturers);
}

// Size1 is for the top level array, size2 for the lower.
void print_2d_array(float** data_in, unsigned int size1, unsigned int size2){
  for (int i = 0; i < size1; i++){
    print_array(data_in[i], size2);
  }
}

void print_2d_int_array(int** data_in, unsigned int size1, unsigned int size2){
  for (int i = 0; i < size1; i++){
    print_int_array(data_in[i], size2);
  }
}

// Print a 2d array represented as a 1d array in 2d format
void print_2d_1d_int_array(int* data_in, unsigned int size1, unsigned int size2){
  for (int i = 0; i < size1; i++)
  {
    printf("[");
    for (int j = 0; j < size2 - 1; j++) 
    {
      printf("%d,", val(data_in, i, j, size2));
    }
    printf("%d]\n", val(data_in, i, size2-1, size2));
  }
}

void print_int_array(int* data_in, unsigned int size) {
  for (int i=0; i < size-1; i++) {
    printf("%d,", data_in[i]);
  }
  printf("%d\n", data_in[size-1]);
}

double sum_array(float* data_in, unsigned int length) {
  double rezult = 0;
  for (int k=0; k < length; k++)
    rezult += data_in[k];
  return rezult;
}



/*__global__ void device_blur_old(float* data_in, float* data_out) {
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const unsigned int y = tid+1;
  const unsigned int stride = y*PADWIDTH;

  extern __shared__ float row[];
  for (int i = stride; i < (stride+PADWIDTH); i++) {
    row[i] = data_in[i];
  }
  __syncthreads();

  #pragma unroll
  for (int x = 1; x <= MATRIX_WIDTH; x++) {
    //for (int y = 1; y <= MATRIX_HEIGHT; y++) {
      data_out[x+stride] = d_avg_of_nine(row, x, y, PADWIDTH);
      //}
  }
  }  */


/* First arg: threads per block,
   Second arg: blocks per grid */
int main(int argc, char** argv)
{
  if (argc < 7) {
      printf("Too few arguments received.\n");
      printf("Usage: %s nthreads nblocks ndays profit_outfile price_outfile loyalty_outfile [seed]\n", argv[0]);
      exit(1);
  }

  int threadsPerBlock = atoi(argv[1]);
  int blocksPerGrid = atoi(argv[2]);
  int days = atoi(argv[3]);
  char* profitFilename = argv[4];
  char* priceFilename = argv[5];
  char* loyaltyFilename = argv[6];

  int devID;
  hipDeviceProp_t props;

  // get number of SMs on this GPU
  cutilSafeCall(hipGetDevice(&devID));
  cutilSafeCall(hipGetDeviceProperties(&props, devID));

  // ---------------------------------------------

  // If more than 7 arguments received, there should be a seed present so use
  // the seed to initialise the random number generator. Otherwise, just use
  // the current time.
  if (argc > 7)
      srand(atoi(argv[7]));
  else 
      srand(time(NULL));
  /* int i; */
  /* for (i = 0; i < 100; ++i) { */
  /*   printf("%lf\n", positive_gaussrand() + 1); */
  /* } */

  // Start of hard work...
  clock_t start_time = clock();
  time_t start, end;
  time(&start);

  int* loyalty = init_loyalty();
  int* marginal_cost = init_marginal_cost();
  int* max_cost = init_max_cost(marginal_cost);
  int* price = init_prices(marginal_cost);
  int* price_strategy = init_strategy();
  profits* profit_history = init_profits();

  host_equilibriate(price, loyalty, profit_history, price_strategy, marginal_cost, max_cost, days, LOYALTY_ENABLED, profitFilename, priceFilename, loyaltyFilename);

  clock_t end_time = clock();
  time(&end);
  
  printf("CPU time taken: %fms\n", (double)(end_time-start_time)/CLOCKS_PER_SEC);
  printf("Wall clock time taken: %d secs\n", (int)(end-start));
  
  

  // allocate host memory 
  /*  unsigned int mem_size = sizeof(float) * PADWIDTH*PADHEIGHT;
  float* h_data_in      = (float*)calloc(sizeof(float), mem_size);
  float* h_data_out     = (float*)malloc(mem_size);

  printf("Input size : x:%d, y:%d\n", MATRIX_WIDTH, MATRIX_HEIGHT);
  //printf("Grid size  : %d\n", GRID_SIZE);
  //  printf("Block size : %d\n", BLOCK_SIZE);

  printf("Grid size  : %d\n", blocksPerGrid);
    printf("Block size : %d\n", threadsPerBlock);


  datainit(h_data_in);

  // allocate device memory
  float* d_data_in;
  cutilSafeCall(hipMalloc((void**) &d_data_in, mem_size));
  float* d_data_out;
  cutilSafeCall(hipMalloc((void**) &d_data_out, mem_size));

  cutilSafeCall(hipMemcpy(d_data_in, h_data_in, 
			   mem_size, hipMemcpyHostToDevice));

  unsigned int globalOffset = 0;
  unsigned int rowOffset = globalOffset % MATRIX_WIDTH;
  
  // set up kernel for execution
  unsigned int timerd = 0;
  cutilCheckError(cutCreateTimer(&timerd));
  cutilCheckError(cutStartTimer(timerd));  
  
  //printf("For loop: \n");
  for (int j = 0; j < ITERS; j++) {
    //printf("%d, ",j);
    cutilSafeCall(hipMemset(d_data_out, 0, mem_size));
   
    //int shmemSize = PADWIDTH*threadsPerBlock*sizeof(float);

    device_blur<<<blocksPerGrid, threadsPerBlock>>>(d_data_in, d_data_out, globalOffset, rowOffset);
    hipDeviceSynchronize();
    
    //cutilSafeCall(hipFree(d_data_in));
  }
  
  // check if kernel execution generated an error
  cutilCheckMsg("Kernel execution failed");

  // stop and destroy timer
  cutilCheckError(cutStopTimer(timerd));
  double dSeconds = cutGetTimerValue(timerd)/(1000.0);
  double dNumOps = ITERS * MATRIX_WIDTH * MATRIX_HEIGHT * 42;
  double gflops = dNumOps/dSeconds/1.0e9;
  double averageDevTime = dSeconds/ITERS*1000; // milliseconds

  //Log throughput
  printf("Device average exec time: %.8f milliseconds\n", averageDevTime);
  printf("Throughput = %.4f GFlop/s\n", gflops);
  cutilCheckError(cutDeleteTimer(timerd));

  cutilSafeCall(hipMemcpy(h_data_out, d_data_out, 
			   mem_size, hipMemcpyDeviceToHost));

  printf("Run %d Kernels.\n\n", ITERS);
  unsigned int timer = 0;
  cutilCheckError(cutCreateTimer(&timer));
  cutilCheckError(cutStartTimer(timer));  
  
  float* host_blur_data_out = (float*)calloc(sizeof(float),mem_size);  

  for (int j = 0; j < ITERS; j++) {
    host_blur(h_data_in, host_blur_data_out);
  }

  //printf("Before calloc\n");
//  float* fake_blur_data_out = (float*)calloc(sizeof(float),mem_size);  
//  printf("After calloc\n");
//  for (int j = 0; j < ITERS; j++) {
//    fake_device_blur(h_data_in, fake_blur_data_out, 0, 0);
//    }


  // check if kernel execution generated an error
  cutilCheckMsg("Kernel execution failed");

  // stop and destroy timer
  cutilCheckError(cutStopTimer(timer));
  double hSeconds = cutGetTimerValue(timer)/(1000.0);
  double averageTime = hSeconds/ITERS; // milliseconds
  printf("Host average exec time: %.8f milliseconds\n", averageTime*1000);

  double sumOfHostBlur = sum_array(host_blur_data_out, PADWIDTH*PADHEIGHT);
  double sumOfDevBlur = sum_array(h_data_out, PADWIDTH*PADHEIGHT);
  printf("HostBlur result: %.4f\n", sumOfHostBlur);
  printf("DeviceBlur result: %.4f\n", sumOfDevBlur);

  //double sumOfFakeBlur = sum_array(fake_blur_data_out, PADWIDTH*PADHEIGHT);
  //printf("FakeBlur result: %.4f\n", sumOfFakeBlur);

  printf("IN MATRIX:\n");
  print_matrix(h_data_in);
  printf("DEV OUT MATRIX:\n");
  print_matrix(h_data_out);
  printf("HOST OUT MATRIX:\n");
  print_matrix(host_blur_data_out);
  printf("FAKE OUT MATRIX:\n");
  print_matrix(fake_blur_data_out);

  // clean up memory
  free(h_data_in);
  free(h_data_out);
  free(host_blur_data_out);
  //free(fake_blur_data_out);
  
  // exit and clean up device status
  hipDeviceReset();*/
}
